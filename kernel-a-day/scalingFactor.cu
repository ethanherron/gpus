#include <iostream>
#include <hip/hip_runtime.h>

#define N 16

// kernel for scaling a matrix A by a value factor
__global__ void scalingFactor(float* A, float factor, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int idx = row * n + col;
        A[idx] = A[idx] * factor;
    }
}


int main() {
    // allocate and init host mem
    size_t size = N * N * sizeof(float);
    float* h_A = (float*)malloc(size);
    float h_factor = 2.0f;

    // simple init by filling a and b with 1s
    for (int i = 0; i < N * N; i++) {
        h_A[i] = (i % 2 == 0) ? (float)i : -(float)i;
    }

    // allocate device mem
    float* d_A = nullptr;
    hipMalloc(&d_A, size);

    // copy host data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    // config and launch kernel
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x,
                 (N + blockDim.y -1) / blockDim.y);
    scalingFactor<<<gridDim, blockDim>>>(d_A, h_factor, N);
    hipDeviceSynchronize();

    // copy results back to host from device
    hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);

    // print results
    std::cout << "After scaling (row 1):  " << std::endl;
    for (int i = 0; i < N; i++) {
        std::cout << h_A[i] << " ";
    }
    std::cout << std::endl;

    // clean up on this disaster
    free(h_A);
    hipFree(d_A);

    return 0;
}
